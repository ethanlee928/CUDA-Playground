
#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void cudaHelloWorld() {
  int bid = blockIdx.x;
  int tid = threadIdx.x;
  printf("Hello World from block %d and thread %d\n", bid, tid);
}

__global__ void cuda3DHelloWorld() {
  int bdx = blockDim.x, bdy = blockDim.y;
  int bx = blockIdx.x, by = blockIdx.y, bz = blockIdx.z;
  int tx = threadIdx.x, ty = threadIdx.y, tz = threadIdx.z;
  int tid = tz * bdx * bdy + ty * bdx + tx;
  printf("[%d] Hello World from block (%d, %d, %d), thread (%d, %d, %d) \n",
         tid, bx, by, bz, tx, ty, tz);
}

int main() {
  cudaHelloWorld<<<3, 5>>>();
  hipDeviceSynchronize();

  printf("\n3D Block Thread\n");
  const dim3 gridSize(3, 3, 3);
  const dim3 blockSize(3, 3, 3);
  cuda3DHelloWorld<<<gridSize, blockSize>>>();
  hipDeviceSynchronize();
  return 0;
}
