
#include <hip/hip_runtime.h>
#include <stdio.h>

void helloWorld() { printf("Hello World!\n"); }

__global__ void cudaHelloWorld() { printf("Hello World from GPU!\n"); }

int main(void) {
  helloWorld();
  cudaHelloWorld<<<1, 1>>>();
  hipDeviceSynchronize();
  return 0;
}
