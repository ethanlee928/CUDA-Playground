#include "hip/hip_runtime.h"
#include "vectorAdd.h"

#define N 10000000

__global__ void vectorAdd(float *out, float *x1, float *x2, int n) {
  int tid = blockDim.x * blockIdx.x + threadIdx.x;
  if (tid < n) {
    out[tid] = x1[tid] + x2[tid];
  }
}

int main() {
  size_t SIZE = sizeof(float) * N;

  float *out = (float *)malloc(SIZE);
  float *x1 = (float *)malloc(SIZE);
  float *x2 = (float *)malloc(SIZE);
  float *d_out, *d_x1, *d_x2;

  hipMalloc((void **)&d_out, SIZE);
  hipMalloc((void **)&d_x1, SIZE);
  hipMalloc((void **)&d_x2, SIZE);

  for (int i = 0; i < N; i++) {
    x1[i] = i;
    x2[i] = i * i;
  }

  hipMemcpy(d_x1, x1, SIZE, hipMemcpyHostToDevice);
  hipMemcpy(d_x2, x2, SIZE, hipMemcpyHostToDevice);

  int blockSize = 500;
  int gridSize = N / 500 + 1;
  vectorAdd<<<gridSize, blockSize>>>(d_out, d_x1, d_x2, N);
  hipFree(d_x1);
  hipFree(d_x2);
  hipMemcpy(out, d_out, SIZE, hipMemcpyDeviceToHost);
  hipFree(d_out);

  testCase(out, x1, x2, N);
  return 0;
}
